#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "ScanHost.cu.h"


int scanIncTest(bool is_segmented) {
    const unsigned int num_threads = 8353455;
    const unsigned int block_size  = 512;
    unsigned int mem_size = num_threads * sizeof(int);

    int* h_in    = (int*) malloc(mem_size);
    int* h_out   = (int*) malloc(mem_size);
    int* flags_h = (int*) malloc(num_threads*sizeof(int));

    int sgm_size = 123;
    { // init segments and flags
        for(unsigned int i=0; i<num_threads; i++) {
            h_in   [i] = 1; 
            flags_h[i] = (i % sgm_size == 0) ? 1 : 0;
        }
    }

    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL); 


    { // calling inclusive (segmented) scan
        int* d_in;
        int* d_out;
        int* flags_d;
        hipMalloc((void**)&d_in ,   mem_size);
        hipMalloc((void**)&d_out,   mem_size);
        hipMalloc((void**)&flags_d, num_threads*sizeof(int));

        // copy host memory to device
        hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);
        hipMemcpy(flags_d, flags_h, num_threads*sizeof(int), hipMemcpyHostToDevice);

        // execute kernel
        if(is_segmented)
            sgmScanInc< Add<int>,int > ( block_size, num_threads, d_in, flags_d, d_out );
        else
            scanInc< Add<int>,int > ( block_size, num_threads, d_in, d_out );

        // copy host memory to device
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

        // cleanup memory
        hipFree(d_in );
        hipFree(d_out);
        hipFree(flags_d);
    }

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
    printf("Scan Inclusive GPU Kernel runs in: %lu microsecs\n", elapsed);

    // validation
    bool success = true;
    int  accum   = 0;
    if(is_segmented) {
        for(int i=0; i<num_threads; i++) {
            if (i % sgm_size == 0) accum  = 0;
            accum += 1;
            
            if ( accum != h_out[i] ) { 
                success = false;
                //printf("Scan Inclusive Violation: %.1d should be %.1d\n", h_out[i], accum);
            }
        }        
    } else {
        for(int i=0; i<num_threads; i++) {
            accum += 1;
 
            if ( accum != h_out[i] ) { 
                success = false;
                //printf("Scan Inclusive Violation: %.1d should be %.1d\n", h_out[i], accum);
            }
        }        
    }

    if(success) printf("\nScan Inclusive +   VALID RESULT!\n");
    else        printf("\nScan Inclusive + INVALID RESULT!\n");


    // cleanup memory
    free(h_in );
    free(h_out);
    free(flags_h);

    return 0;
}

int main(int argc, char** argv) {
    scanIncTest(true);
    scanIncTest(true);
    scanIncTest(false);
}
